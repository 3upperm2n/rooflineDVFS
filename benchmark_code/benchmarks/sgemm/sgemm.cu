/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* 
 * Main entry of dense matrix-matrix multiplication kernel
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <iostream>
#include "parboil.h"
#include "sgemm_kernel.cu"

// I/O routines
bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float>&v);
bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float>&);

int
main (int argc, char *argv[]) {

  //struct pb_TimerSet timers;

  float *dA, *dB, *dC;
  size_t A_sz, B_sz, C_sz;
  int matArow, matAcol;
  int matBrow, matBcol;
  std::vector<float> matA, matBT;

  //pb_InitializeTimerSet(&timers);

  /* Initialize the parameters structure */
  struct pb_Parameters *params = (struct pb_Parameters *)malloc(sizeof(struct pb_Parameters));
  params->outFile = NULL;
  params->inpFiles = (char **)malloc(sizeof(char *)*3);
  params->inpFiles[0] = NULL;

  /* Read command line. Expect 3 inputs: A, B and B^T 
     in column-major layout*/
#ifdef SIZE0
  params->inpFiles[0] = "~/software/parboil-2.5/datasets/sgemm/small/input/matrix1.txt";
  params->inpFiles[1] = "~/software/parboil-2.5/datasets/sgemm/small/input/matrix2.txt";
  params->inpFiles[2] = "~/software/parboil-2.5/datasets/sgemm/small/input/matrix2t.txt";
#endif
#ifdef SIZE1
  params->inpFiles[0] = "~/software/parboil-2.5/datasets/sgemm/medium/input/matrix1.txt";
  params->inpFiles[1] = "~/software/parboil-2.5/datasets/sgemm/medium/input/matrix2.txt";
  params->inpFiles[2] = "~/software/parboil-2.5/datasets/sgemm/medium/input/matrix2t.txt";
#endif
  /* Read in data */
  //pb_SwitchToTimer(&timers, pb_TimerID_IO);

  // load A
  readColMajorMatrixFile(params->inpFiles[0],
      matArow, matAcol, matA);
  // copy A to device memory
  A_sz = matArow*matAcol*sizeof(float);

  // load B^T
  readColMajorMatrixFile(params->inpFiles[2],
      matBcol, matBrow, matBT);

 // pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
  B_sz = matBrow*matBcol*sizeof(float);

  // allocate space for C
  C_sz = matArow*matBcol*sizeof(float);

  // CUDA memory allocation
  std::vector<float> matC(matArow*matBcol);
  hipMalloc((void**)&dA, A_sz);
  hipMalloc((void**)&dB, B_sz);
  hipMalloc((void**)&dC, C_sz);
  
  // Copy A and B^T into device memory
  //pb_SwitchToTimer( &timers, pb_TimerID_COPY );
  hipMemcpy(dA, &matA.front(), A_sz, hipMemcpyHostToDevice); 
  hipMemcpy(dB, &matBT.front(), B_sz, hipMemcpyHostToDevice); 

  //pb_SwitchToTimer( &timers, pb_TimerID_KERNEL );

  // Use standard sgemm interface
  regtileSgemm('N', 'T', matArow, matBcol, matAcol, 1.0f, \
      dA, matArow, dB, matBcol, 0.0f, dC, matArow);

  if (params->outFile) {
    //pb_SwitchToTimer( &timers, pb_TimerID_COPY );
    hipMemcpy(&matC.front(), dC, C_sz, hipMemcpyDeviceToHost);
    /* Write C to file */
    //pb_SwitchToTimer(&timers, pb_TimerID_IO);
    writeColMajorMatrixFile(params->outFile,
	matArow, matBcol, matC); 
  }

  //pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  //double GPUtime = pb_GetElapsedTime(&(timers.timers[pb_TimerID_KERNEL]));
  //std::cout<< "GFLOPs = " << 2.* matArow * matBcol * matAcol/GPUtime/1e9 << std::endl;
  //pb_PrintTimerSet(&timers);
  //pb_FreeParameters(params);
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);
  return 0;
}

/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* I/O routines for reading and writing matrices in column-major
 * layout
 */

#include<fstream>
#include<iostream>
#include<vector>

bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float>&v)
{
  std::cerr << "Opening file:"<< fn << std::endl;
  std::fstream f(fn, std::fstream::in);
  if ( !f.good() ) {
    return false;
  }

  // Read # of rows and cols
  f >> nr_row;
  f >> nr_col;

  float data;
  std::cerr << "Matrix dimension: "<<nr_row<<"x"<<nr_col<<std::endl;
  while (f.good() ) {
    f >> data;
    v.push_back(data);
  }
  v.pop_back(); // remove the duplicated last element

  return true;
}

bool writeColMajorMatrixFile(const char *fn, int nr_row, int nr_col, std::vector<float>&v)
{
  std::cerr << "Opening file:"<< fn << " for write." << std::endl;
  std::fstream f(fn, std::fstream::out);
  if ( !f.good() ) {
    return false;
  }

  // Read # of rows and cols
  f << nr_row << " "<<nr_col<<" ";

  std::cerr << "Matrix dimension: "<<nr_row<<"x"<<nr_col<<std::endl;
  for (unsigned i = 0; i < v.size(); ++i) {
    f << v[i] << ' ';
  }
  f << "\n";
  return true;

}
